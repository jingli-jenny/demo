#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */


#include <faiss/gpu/utils/Float16.cuh>
//#include <faiss/gpu/utils/nvidia/fp16_emu.cuh>
#include <faiss/impl/FaissAssert.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>

#ifdef FAISS_USE_FLOAT16

namespace faiss { namespace gpu {

bool getDeviceSupportsFloat16Math(int device) {
  const auto& prop = getDeviceProperties(device);

  return (prop.major >= 6 ||
          (prop.major == 5 && prop.minor >= 3));
}

__half hostFloat2Half(float a) {
#if CUDA_VERSION >= 9000
  __half_raw raw;
  //raw.x = cpu_float2half_rn(a).x;
  FAISS_ASSERT_FMT(false, "%s", "cpu_float2half_rn() not support");
  return __half(raw);
#else
  __half h;
  //h.x = cpu_float2half_rn(a).x;
  FAISS_ASSERT_FMT(false, "%s", "cpu_float2half_rn() not support");
  return h;
#endif
}

} } // namespace

#endif // FAISS_USE_FLOAT16
